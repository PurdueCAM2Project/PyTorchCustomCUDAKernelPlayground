#include "gorby_sdpa.cuh"
#include "cuda_check.hpp"

#include <hip/hip_runtime.h>
#include <cutlass/gemm/device/gemm.h>

using namespace gorby::utils;

#define CUTLASS_CHECK(status)                                                                    \
	{                                                                                              \
	cutlass::Status error = status;                                                              \
	if (error != cutlass::Status::kSuccess) {                                                    \
		std::cerr << "Got cutlass error: " << cutlassGetStatusString(error) << " at: " << __LINE__ \
				<< std::endl;                                                                    \
		exit(EXIT_FAILURE);                                                                        \
	}                                                                                            \
}

namespace gorby{
    namespace sdpa{
        // CUDA Binding - Simple CUTLASS SGEMM kernel
		// See https://github.com/NVIDIA/cutlass/issues/282 for example
        torch::Tensor cutlass_sgemm_nn(
			torch::Tensor A, 
			torch::Tensor B, 
			torch::Tensor C
		) {
			// Create output tensor D
			auto D_options = torch::TensorOptions()
				.dtype(C.dtype())
				.layout(C.layout())
				.device(C.device())
				.requires_grad(false);

			torch::Tensor D = torch::empty_like(C, D_options);

			// Get Tensor Accessors - assert type is float and A,B,C have 2 dimensions
			auto a = A.packed_accessor64<float, 2>();
			auto b = B.packed_accessor64<float, 2>();
			auto c = C.packed_accessor64<float, 2>();
			auto d = D.packed_accessor64<float, 2>();

			// Let's do the matrix multiplication via CUTLASS
			// Create GEMM instance
			using CutlassSGEMM_NNOperator = cutlass::gemm::device::Gemm<
				float, cutlass::layout::ColumnMajor,
				float, cutlass::layout::ColumnMajor,
				float, cutlass::layout::ColumnMajor,
				float,
				cutlass::arch::OpClassSimt,
				cutlass::arch::Sm86
				// This code section describes the tile size a thread block will compute
				// cutlass::gemm::GemmShape<128, 128, 1>,
				// This code section describes tile size a warp will compute
				// cutlass::gemm::GemmShape<64, 64, 1>,
				// This code section describes the size of MMA op
				// cutlass::gemm::GemmShape<32, 32, 1>
			>;

			CutlassSGEMM_NNOperator cutlass_sgemm_nn_operator_instance;

			// Get Problem Size
			int M = (int) A.size(0);
			int N = (int) A.size(1);
			int K = (int) C.size(1);

			CutlassSGEMM_NNOperator::Arguments args(
				{M, N, K},
				{(float*)A.data_ptr(), cutlass::layout::ColumnMajor(a.stride(0))},
				{(float*)B.data_ptr(), cutlass::layout::ColumnMajor(b.stride(0))},
				{(float*)C.data_ptr(), cutlass::layout::ColumnMajor(c.stride(0))},
				{(float*)D.data_ptr(), cutlass::layout::ColumnMajor(d.stride(0))},
				{1.0f, 1.0f}
			);
			
			// Invoke the CUTLASS GEMM template
			cutlass::Status status = cutlass_sgemm_nn_operator_instance(args);
			CUTLASS_CHECK(status);

			// Return!
			return D;
		}


        // Definitions
        torch::Tensor gorby_sdpa_forward(
        	torch::Tensor A, torch::Tensor B, torch::Tensor C
        ) {
			CHECK_CUDA(A);
			CHECK_CUDA(B);
			CHECK_CUDA(C);
			CHECK_CONTIGUOUS(A);
			CHECK_CONTIGUOUS(B);
			CHECK_CONTIGUOUS(C);
			CHECK_SAME_TYPE(A, B);
			CHECK_SAME_TYPE(B, C);
			TORCH_CHECK(A.size(1) == B.size(0) && C.size(0) == A.size(0) && C.size(1) == B.size(1));

			torch::Tensor D = cutlass_sgemm_nn(A, B, C);

			return D;
		}
    }
}